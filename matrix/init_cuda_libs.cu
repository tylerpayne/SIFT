#include <matrix.h>

#ifdef __cplusplus
extern "C" {
#endif

hipblasHandle_t _cublasHandle;
hiprandGenerator_t _curandGenerator;

int init_cuda_libs()
{

  //cublas
  cublas_safe_call(
    hipblasCreate(&_cublasHandle)
  );

  //hiprand
  curand_safe_call(
    hiprandCreateGenerator(
      &_curandGenerator, HIPRAND_RNG_PSEUDO_DEFAULT
    )
  );
  curand_safe_call(
    hiprandSetPseudoRandomGeneratorSeed(
      _curandGenerator, 1234ULL
    )
  );

  return 0;
}


#ifdef __cplusplus
}
#endif
