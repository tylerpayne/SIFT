#include <matrix.h>
#include <hiprand.h>

#ifdef __cplusplus
extern "C" {
#endif

int uniform(Matrix *out, Shape shape)
{
  memassert(out,DEVICE);
  float *dev_ptr;
  cuda_safe_call(hipMalloc(&dev_ptr,shape.width*shape.height*sizeof(float)));
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen,
                HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen,
                1234ULL);
  hiprandGenerateUniform(gen, dev_ptr, shape.width*shape.height);
  out->dev_ptr = dev_ptr;
  return 0;
}

#ifdef __cplusplus
}
#endif
