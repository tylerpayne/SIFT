#include <chai.h>

namespace chai {
namespace cuda {
extern void cublas_safe_call(hipblasStatus_t err);
extern void curand_safe_call(hiprandStatus_t err);

hipStream_t _cudaStream;
hipblasHandle_t _cublasHandle;
hiprandGenerator_t _curandGenerator;

void init_cuda_libs() {

  _cudaStream = 0;

  // cublas
  cublas_safe_call(hipblasCreate(&_cublasHandle));

  // hiprand
  curand_safe_call(
      hiprandCreateGenerator(&_curandGenerator, HIPRAND_RNG_PSEUDO_DEFAULT));

  curand_safe_call(
      hiprandSetPseudoRandomGeneratorSeed(_curandGenerator, 1234ULL));
}
}
}
