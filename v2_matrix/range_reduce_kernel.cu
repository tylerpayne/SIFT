#include "hip/hip_runtime.h"
#include <matrix.h>

#ifdef __cplusplus
extern "C" {
#endif

__global__ void range_reduce_kernel(float *a, float from, float to, int *d_index, Shape shape)
{
  extern __shared__ int results[];
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  if (x<shape.width*shape.height)
  {
    if ((a[x]-to) < -0.01 && (a[x]-from) > -0.01)
    {
      results[threadIdx.x] = x;
    } else
    {
      results[threadIdx.x] = 0;
    }
    __syncthreads();
    if (threadIdx.x == 0)
    {
      int counter = 1;
      for (int i = 0; i < blockDim.x; i++)
      {
        if (results[i] != 0)
          d_index[blockDim.x*blockIdx.x + counter++] = results[i];
      }
      d_index[blockDim.x*blockIdx.x] = counter;
    }
  }
}

#ifdef __cplusplus
}
#endif
