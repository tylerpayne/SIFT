#include "hip/hip_runtime.h"
#include "MatrixUtil.h"
#include "MathKernels.cu"

hipError_t cudaErr;
hipblasStatus_t cublasStat;

void cublasErrCheck(hipblasStatus_t stat)
{
  if (stat != HIPBLAS_STATUS_SUCCESS)
  {
    printf("\nCUBLAS ERR: %i\n",stat);
  }
}

void freeCudaMatrixDeviceMemory(Matrix* mat)
{
  hipFree(mat->devicePtr);
}

void copyDeviceToDeviceCudaMatrix(MatrixUtil* self, Matrix* A, Matrix* B)
{
    hipSetDevice(self->deviceId);
    size_t size = sizeof(float)*A->shape[0]*A->shape[1];
    hipMemcpyAsync(B->devicePtr,A->devicePtr,size,hipMemcpyDeviceToDevice,self->stream);
}

void copyHostToDeviceCudaMatrix(MatrixUtil* self, Matrix* mat)
{
    hipSetDevice(self->deviceId);
    size_t size = sizeof(float)*mat->shape[0]*mat->shape[1];
    if (VERBOSITY > 3)
    {
      printf("\n\n##### GPU WARNING #####\n");
      printf("Copying from Host to Device");
      printf("\n###################\n\n");
    }
    hipMemcpyAsync(mat->devicePtr,mat->hostPtr,size,hipMemcpyHostToDevice,self->stream);
    mat->isHostSide = 0;
}

void copyDeviceToHostCudaMatrix(MatrixUtil* self, Matrix* mat)
{
  hipSetDevice(self->deviceId);
  if (VERBOSITY > 3)
  {
    printf("\n\n### GPU WARNING ###\n");
    printf("Copying Device to Host");
    printf("\n###################\n\n");
  }
  size_t size = sizeof(float)*mat->shape[0]*mat->shape[1];
  hipMemcpyAsync(mat->hostPtr,mat->devicePtr,size,hipMemcpyDeviceToHost,self->stream);
  mat->isHostSide = 1;
}

float getCudaMatrixElementImpl(Matrix* self, int i, int  j)
{
  if (!self->isHostSide)
  {
    printf("\n### GPU WARNING ###\n");
    printf("Matrix was on device when trying to get.");
    printf("\n###################\n");
  }
  return self->hostPtr[IDX2C(i,j,self->shape[1])];
}

float* getCudaMatrixRegionImpl(Matrix* self, int i, int j, int h, int w)
{
  float* data = (float*)malloc(sizeof(float)*h*w);

  for (int y = 0; y < h; y++)
  {
    for (int x = 0; x < w; x++)
    {
      data[IDX2C(y,x,w)] = self->getElement(self,y+i,x+j);
    }
  }
  return data;
}

void setCudaMatrixElementImpl(Matrix* self, int i, int  j, float x)
{
  if (!self->isHostSide)
  {
    printf("\n### GPU WARNING ###\n");
    printf("Matrix was on device when trying to set!\n");
    printf("\n###################\n");
  }
  self->hostPtr[IDX2C(i,j,self->shape[1])] = x;
}

void setCudaMatrixRegionImpl(Matrix* self, int i, int j, int r, int c, float* data)
{
  int counter = 0;
  for (int z = i; z < i+r; z++)
  {
    for (int y = j; y < j+c; y++)
    {
      self->setElement(self,z,y,data[counter]);
      counter++;
    }
  }
}

Matrix* newEmptyCudaMatrixImpl(int width, int height)
{
  Matrix* m = (Matrix*)malloc(sizeof(Matrix));
  float* h_data = (float*)malloc(sizeof(float)*width*height);
  memset(h_data,0,sizeof(float)*width*height);
  float* d_data;
  hipMalloc(&d_data,sizeof(float)*width*height);
  int* shape = (int*)malloc(sizeof(int)*2);
  shape[0] = height;
  shape[1] = width;

  m->shape = shape;
  m->hostPtr = h_data;
  m->devicePtr = d_data;
  m->isHostSide = 1;
  m->T = HIPBLAS_OP_N;
  m->getElement = getCudaMatrixElementImpl;
  m->getRegion = getCudaMatrixRegionImpl;
  m->setElement = setCudaMatrixElementImpl;
  m->setRegion = setCudaMatrixRegionImpl;
  return m;
}

Matrix* newCudaMatrixImpl(float* data, int width, int height)
{
  Matrix* m = newEmptyCudaMatrixImpl(width,height);
  free(m->hostPtr);
  m->hostPtr = data;
  return m;
}
//############ BEGIN MATH FUNCS ##################
//ADDITION
void addCudaMatrixImpl(MatrixUtil* self, Matrix* A, Matrix* B, Matrix* C)
{
  if (A->isHostSide)
  {
    copyHostToDeviceCudaMatrix(self,A);
  }
  if (B->isHostSide)
  {
    copyHostToDeviceCudaMatrix(self,B);
  }
  if (C->isHostSide)
  {
    copyHostToDeviceCudaMatrix(self,C);
  }
  copyDeviceToDeviceCudaMatrix(self,B,C);
  //hipblasSetStream(self->cublasHandle,self->stream);
  float a = 1;
  cublasErrCheck(hipblasSaxpy(self->cublasHandle,A->shape[0]*A->shape[1],&a,A->devicePtr,1,C->devicePtr,1));
}

//SUBTRACTION
void subtractCudaMatrixImpl(MatrixUtil* self, Matrix* A, Matrix* B, Matrix* C)
{
  if (A->isHostSide)
  {
    copyHostToDeviceCudaMatrix(self,A);
  }
  if (B->isHostSide)
  {
    copyHostToDeviceCudaMatrix(self,B);
  }
  if (C->isHostSide)
  {
    copyHostToDeviceCudaMatrix(self,C);
  }
  copyDeviceToDeviceCudaMatrix(self,A,C);
  //hipblasSetStream(self->cublasHandle,self->stream);
  float a = -1;
  cublasErrCheck(hipblasSaxpy(self->cublasHandle,A->shape[0]*A->shape[1],&a,B->devicePtr,1,C->devicePtr,1));
}

//MULTIPLYCONST
void multiplyConstCudaMatrixImpl(MatrixUtil* self, Matrix* A, float b, Matrix* C)
{
  if (A->isHostSide)
  {
    copyHostToDeviceCudaMatrix(self,A);
  }
  if (C->isHostSide)
  {
    copyHostToDeviceCudaMatrix(self,C);
  }
  //hipblasSetStream(self->cublasHandle,self->stream);
  cublasErrCheck(hipblasSaxpy(self->cublasHandle,A->shape[0]*A->shape[1],&b,A->devicePtr,1,C->devicePtr,1));
}

//DOT
void dotCudaMatrixImpl(MatrixUtil* self, Matrix* A, Matrix* B, Matrix* C)
{
  if (A->isHostSide)
  {
    copyHostToDeviceCudaMatrix(self,A);
  }
  if (B->isHostSide)
  {
    copyHostToDeviceCudaMatrix(self,B);
  }
  if (C->isHostSide)
  {
    copyHostToDeviceCudaMatrix(self,C);
  }
  //hipblasSetStream(self->cublasHandle,self->stream);
  float alpha = 1;
  float beta = 0;
  cublasErrCheck(hipblasSgemm(self->cublasHandle,A->T,B->T,A->shape[0],B->shape[1],A->shape[1],&alpha,A->devicePtr,A->shape[0],B->devicePtr,B->shape[0],&beta,C->devicePtr,C->shape[0]));
}

void SetCUDAMatrixUtilStream(MatrixUtil* self, hipStream_t stream)
{
    self->stream = stream;
    hipblasSetStream(self->cublasHandle, stream);
}

void SetCUDAMatrixUtilDevice(MatrixUtil* self, int device)
{
  self->deviceId = device;
  hipSetDevice(device);

  hipblasHandle_t cublasHandle;
  hipblasCreate(&cublasHandle);
  self->cublasHandle = cublasHandle;

  hipStream_t stream;
  hipStreamCreate(&stream);
  SetCUDAMatrixUtilStream(self,stream);
}

void pprintCudaMatrixImpl(MatrixUtil* self, Matrix* A, char* label)
{
  printf("\n\n################################################");
  printf("\n%s:\n\n",label);
  copyDeviceToHostCudaMatrix(self,A);
  for (int i = 0; i < A->shape[0]; i++)
  {
    for (int j = 0;j< A->shape[1];j++)
    {
      printf("[ %f ]",A->getElement(A,i,j));
    }
    //printf("|  %f",y->getElement(y,i,0));
    printf("\n");
  }
  printf("\n################################################\n\n");
}

MatrixUtil* GetCUDAMatrixUtil(int device)
{
  MatrixUtil* cudaMatrixUtil = (MatrixUtil*)malloc(sizeof(MatrixUtil));
  SetCUDAMatrixUtilDevice(cudaMatrixUtil,device);

  cudaMatrixUtil->newEmptyMatrix = newEmptyCudaMatrixImpl;
  cudaMatrixUtil->pprint = pprintCudaMatrixImpl;
  cudaMatrixUtil->newMatrix = newCudaMatrixImpl;
  cudaMatrixUtil->add = addCudaMatrixImpl;
  cudaMatrixUtil->subtract = subtractCudaMatrixImpl;
  cudaMatrixUtil->dot = dotCudaMatrixImpl;
  cudaMatrixUtil->multiplyConst = multiplyConstCudaMatrixImpl;
/*
  cudaMatrixUtil->multiply = multiplyCudaMatrixImpl;
  cudaMatrixUtil->divide = divideCudaMatrixImpl;
  cudaMatrixUtil->divideConst = divideConstCudaMatrixImpl;
  cudaMatrixUtil->pow = powCudaMatrixImpl;
  cudaMatrixUtil->convolve = convolveCudaMatrixImpl;
  cudaMatrixUtil->sqrt = sqrtCudaMatrixImpl;
  cudaMatrixUtil->exp = expCudaMatrixImpl;
  cudaMatrixUtil->log = logCudaMatrixImpl;
  cudaMatrixUtil->arctan = arctanCudaMatrixImpl;
  cudaMatrixUtil->transpose = transposeCudaMatrixImpl;

  cudaMatrixUtil->inv = invCudaMatrixImpl;
  cudaMatrixUtil->solve = solveCudaMatrixImpl;
  cudaMatrixUtil->lstsq = lstsqCudaMatrixImpl;

  //cudaMatrixUtil->ceil = ceilCudaMatrixImpl;
  //cudaMatrixUtil->floor = floorCudaMatrixImpl;
  //cudaMatrixUtil->abs = absCudaMatrixImpl;
  cudaMatrixUtil->isEqual = isEqualCudaMatrixImpl;
  cudaMatrixUtil->convolve = convolveCudaMatrixImpl;*/
//TODO: cross

  return cudaMatrixUtil;
}
