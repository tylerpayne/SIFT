#include <matrix.h>

#ifdef __cplusplus
extern "C" {
#endif

extern hipStream_t _cudaStream;

int add(Matrix *a, Matrix *b, Matrix *out)
{
  memassert(a,DEVICE);
  memassert(b,DEVICE);

  Matrix *ret;
  if (out != NULL) ret = out;
  else ret = a;

  memassert(ret,DEVICE);

  dim3 bdim,gdim;
  make_launch_parameters(a->shape,1,&bdim,&gdim);
  add_kernel<<<gdim,bdim,0,_cudaStream>>>(a->dev_ptr,b->dev_ptr,ret->dev_ptr,a->shape);
  return 0;
}

int addc(Matrix *a, float b, Matrix *out)
{
  memassert(a,DEVICE);

  Matrix *ret;
  if (out != NULL) ret = out;
  else ret = a;

  memassert(ret,DEVICE);


  dim3 bdim,gdim;
  make_launch_parameters(a->shape,1,&bdim,&gdim);
  addc_kernel<<<gdim,bdim,0,_cudaStream>>>(a->dev_ptr,b,ret->dev_ptr,a->shape);
  return 0;
}

int divide(Matrix *a, Matrix *b, Matrix *out)
{
  memassert(a,DEVICE);
  memassert(b,DEVICE);

  Matrix *ret;
  if (out != NULL) ret = out;
  else ret = a;

  memassert(ret,DEVICE);


  dim3 bdim,gdim;
  make_launch_parameters(a->shape,1,&bdim,&gdim);
  divide_kernel<<<gdim,bdim,0,_cudaStream>>>(a->dev_ptr,b->dev_ptr,ret->dev_ptr,a->shape);
  return 0;
}

int dividec(Matrix *a, float b, Matrix *out)
{
  memassert(a,DEVICE);

  Matrix *ret;
  if (out != NULL) ret = out;
  else ret = a;

  memassert(ret,DEVICE);

  dim3 bdim,gdim;
  make_launch_parameters(a->shape,1,&bdim,&gdim);
  dividec_kernel<<<gdim,bdim,0,_cudaStream>>>(a->dev_ptr,b,ret->dev_ptr,a->shape);
  return 0;
}

int multiply(Matrix *a, Matrix *b, Matrix *out)
{
  memassert(a,DEVICE);
  memassert(b,DEVICE);

  Matrix *ret;
  if (out != NULL) ret = out;
  else ret = a;

  memassert(ret,DEVICE);


  dim3 bdim,gdim;
  make_launch_parameters(a->shape,1,&bdim,&gdim);
  multiply_kernel<<<gdim,bdim,0,_cudaStream>>>(a->dev_ptr,b->dev_ptr,ret->dev_ptr,a->shape);
  return 0;
}

int multiplyc(Matrix *a, float b, Matrix *out)
{
  memassert(a,DEVICE);

  Matrix *ret;
  if (out != NULL) ret = out;
  else ret = a;

  memassert(ret,DEVICE);

  dim3 bdim,gdim;
  make_launch_parameters(a->shape,1,&bdim,&gdim);
  multiplyc_kernel<<<gdim,bdim,0,_cudaStream>>>(a->dev_ptr,b,ret->dev_ptr,a->shape);
  return 0;
}

int subtract(Matrix *a, Matrix *b, Matrix *out)
{
  memassert(a,DEVICE);
  memassert(b,DEVICE);

  Matrix *ret;
  if (out != NULL) ret = out;
  else ret = a;

  memassert(ret,DEVICE);

  dim3 bdim,gdim;
  make_launch_parameters(a->shape,1,&bdim,&gdim);
  subtract_kernel<<<gdim,bdim,0,_cudaStream>>>(a->dev_ptr,b->dev_ptr,ret->dev_ptr,a->shape);
  return 0;
}

int subtractc(Matrix *a, float b, Matrix *out)
{
  memassert(a,DEVICE);

  Matrix *ret;
  if (out != NULL) ret = out;
  else ret = a;

  memassert(ret,DEVICE);

  dim3 bdim,gdim;
  make_launch_parameters(a->shape,1,&bdim,&gdim);
  subtractc_kernel<<<gdim,bdim,0,_cudaStream>>>(a->dev_ptr,b,ret->dev_ptr,a->shape);
  return 0;
}

#ifdef __cplusplus
}
#endif
