#include <matrix.h>

#ifdef __cplusplus
extern "C" {
#endif

extern hipblasHandle_t _cublasHandle;

int euclid_norm(Matrix *a, float *out)
{
  memassert(a,DEVICE);
  cublas_safe_call(
    hipblasSnrm2(_cublasHandle,
      SHAPE2LEN(a->shape),a->dev_ptr, 1,
      out
    )
  );
  return 0;
}

#ifdef __cplusplus
}
#endif
