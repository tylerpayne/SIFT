#include <chai.h>

namespace chai {
namespace cuda {
void cuda_safe_call(hipError_t err) {
  if (err != hipSuccess)
    printf("CUDA Error: %s\n", hipGetErrorString(err));
  // exit(1);
}

void cublas_safe_call(hipblasStatus_t err) {
  if (err != HIPBLAS_STATUS_SUCCESS)
    printf("CUBLAS ERROR: %i\n", err);
  // exit(1);
}

void curand_safe_call(hiprandStatus_t err) {
  if (err != HIPRAND_STATUS_SUCCESS)
    printf("CURAND ERROR: %i\n", err);
  // exit(1);
}

template <> void safe_call<hipError_t>(hipError_t err) {
  cuda_safe_call(err);
}

template <> void safe_call<hipblasStatus_t>(hipblasStatus_t err) {
  cublas_safe_call(err);
}

template <> void safe_call<hiprandStatus_t>(hiprandStatus_t err) {
  curand_safe_call(err);
}
}
}
