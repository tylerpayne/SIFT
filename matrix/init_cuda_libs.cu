#include <matrix.h>

#ifdef __cplusplus
extern "C" {
#endif

hipStream_t _cudaStream;
hipblasHandle_t _cublasHandle;
hiprandGenerator_t _curandGenerator;

int init_cuda_libs()
{

  _cudaStream = 0;

  //cublas
  cublas_safe_call(
    hipblasCreate(&_cublasHandle)
  );

  //hiprand
  curand_safe_call(
    hiprandCreateGenerator(
      &_curandGenerator, HIPRAND_RNG_PSEUDO_DEFAULT
    )
  );

  curand_safe_call(
    hiprandSetPseudoRandomGeneratorSeed(
      _curandGenerator, 1234ULL
    )
  );

  return 0;
}


#ifdef __cplusplus
}
#endif
