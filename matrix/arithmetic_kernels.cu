#include "hip/hip_runtime.h"
#include <matrix.h>

#ifdef __cplusplus
extern "C" {
#endif

__global__ void add_kernel(float *a, float *b, float *c, Shape shape)
{
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  if (x<shape.width*shape.height)
  {
    c[x] = a[x] + b[x];
  }
}

__global__ void divide_kernel(float *a, float *b, float *c, Shape shape)
{
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  if (x<shape.width*shape.height)
  {
    c[x] = a[x] / b[x];
  }
}

__global__ void multiply_kernel(float *a, float *b, float *c, Shape shape)
{
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  if (x<shape.width*shape.height)
  {
    c[x] = a[x] * b[x];
  }
}

__global__ void subtract_kernel(float *a, float *b, float *c, Shape shape)
{
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  if (x<shape.width*shape.height)
  {
    c[x] = a[x] - b[x];
  }
}

#ifdef __cplusplus
}
#endif
