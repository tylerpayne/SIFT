#include <chai.h>

namespace chai
{
  namespace cuda
  {
    extern void cublas_safe_call(hipblasStatus_t err);
    extern void curand_safe_call(hiprandStatus_t err);

    extern hipblasHandle_t _cublasHandle;
    extern hiprandGenerator_t _curandGenerator;

    void destroy_cuda_libs()
    {
      //cublas
      cublas_safe_call(hipblasDestroy(_cublasHandle));

      //hiprand
      curand_safe_call(hiprandDestroyGenerator(_curandGenerator));
    }

  }
}
