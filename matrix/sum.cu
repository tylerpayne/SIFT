#include <matrix.h>

#ifdef __cplusplus
extern "C" {
#endif

extern hipblasHandle_t _cublasHandle;

int sum(Matrix *a, float *out)
{
  memassert(a,DEVICE);
  cublas_safe_call(
    hipblasSasum(_cublasHandle,
      SHAPE2LEN(a->shape),a->dev_ptr, 1,
      out
    )
  );
  return 0;
}

#ifdef __cplusplus
}
#endif
