#include "hip/hip_runtime.h"
#include <matrix.h>

template<typename K>
__global__ void sqrt_kernel(K *a, K *b, Shape shape);

template<typename K>
__global__ void abs_kernel(K *a, K *b, Shape shape);

template<typename K>
__global__ void exp_kernel(K *a, K *b, Shape shape);

template<typename K>
__global__ void log_kernel(K *a, K *b, Shape shape);

template<typename K>
__global__ void pow_kernel(K *a, K b, K *c, Shape shape);

template<>
__global__ void sqrt_kernel<float>(float *a, float *b, Shape shape)
{
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  if (x<shape.width*shape.height)
  {
    b[x] = sqrtf(a[x]);
  }
}

template<>
__global__ void abs_kernel<float>(float *a, float *b, Shape shape)
{
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  if (x<shape.width*shape.height)
  {
    b[x] = fabsf(a[x]);
  }
}

template
__global__ void exp_kernel<float>(float *a, float *b, Shape shape)
{
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  if (x<shape.width*shape.height)
  {
    b[x] = expf(a[x]);
  }
}

template
__global__ void log_kernel<float>(float *a, float *b, Shape shape)
{
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  if (x<shape.width*shape.height)
  {
    b[x] = logf(a[x]);
  }
}

template<>
__global__ void pow_kernel<float>(float *a, float b, float *c, Shape shape)
{
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  if (x<shape.width*shape.height)
  {
    c[x] = powf(a[x],b);
  }
}
