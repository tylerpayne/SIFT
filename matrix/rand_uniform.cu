#include <matrix.h>

#ifdef __cplusplus
extern "C" {
#endif

extern hiprandGenerator_t _curandGenerator;

int rand_uniform(Matrix *out)
{
  memassert(out,DEVICE);
  float *dev_ptr;
  Shape shape = out->shape;
  cuda_safe_call(
    hipMalloc(&dev_ptr,
      shape.width*shape.height*sizeof(float)
    )
  );

  curand_safe_call(
    hiprandGenerateUniform(_curandGenerator,
       dev_ptr, shape.width*shape.height
     )
   );

  out->dev_ptr = dev_ptr;
  return 0;
}

#ifdef __cplusplus
}
#endif
