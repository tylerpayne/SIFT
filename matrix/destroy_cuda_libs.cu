#include <matrix.h>

#ifdef __cplusplus
extern "C" {
#endif

extern hipblasHandle_t _cublasHandle;
extern hiprandGenerator_t _curandGenerator;

int destroy_cuda_libs()
{
  //cublas
  cublas_safe_call(hipblasDestroy(_cublasHandle));

  //hiprand
  curand_safe_call(hiprandDestroyGenerator(_curandGenerator));

  return 0;
}


#ifdef __cplusplus
}
#endif
