#include <hip/hip_runtime.h>

__device__ int IDX2CKernel(int i, int j, int td)
{
  return (i*td)+j;
}

__global__ void MatAdd(float* A, float* B, float* C,int ld, int td)
{
  int col = (blockIdx.y * blockDim.y) + threadIdx.y;
  int row = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (row < ld && col < td)
  {
    C[IDX2CKernel(row,col,td)] = A[IDX2CKernel(row,col,td)] + B[IDX2CKernel(row,col,td)];
  }
}
//ADDITION
__global__ void MatAddConst(float* A, float B, float* C,int ld, int td)
{
  int col = (blockIdx.y * blockDim.y) + threadIdx.y;
  int row = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (row < ld && col < td)
  {
    C[IDX2CKernel(row,col,td)] = A[IDX2CKernel(row,col,td)] + B;
  }
}
//SUBTRACTION
__global__ void MatSubtract(float* A, float* B, float* C,int ld, int td)
{
  int col = (blockIdx.y * blockDim.y) + threadIdx.y;
  int row = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (row < ld && col < td)
  {
    C[IDX2CKernel(row,col,td)] = A[IDX2CKernel(row,col,td)] - B[IDX2CKernel(row,col,td)];
  }
}

__global__ void MatSubtractConst(float* A, float B, float* C, int ld, int td)
{
  int col = (blockIdx.y * blockDim.y) + threadIdx.y;
  int row = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (row < ld && col < td)
  {
    C[IDX2CKernel(row,col,td)] = A[IDX2CKernel(row,col,td)] - B;
  }
}

//MULTIPLICATION
__global__ void MatMult(float* A, float* B, float* C, int ld, int td)
{
  int col = (blockIdx.y * blockDim.y) + threadIdx.y;
  int row = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (row < ld && col < td)
  {
    C[IDX2CKernel(row,col,td)] = A[IDX2CKernel(row,col,td)] * B[IDX2CKernel(row,col,td)];
  }
}

__global__ void MatMultConst(float* A, float B, float* C, int ld, int td)
{
  int col = (blockIdx.y * blockDim.y) + threadIdx.y;
  int row = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (row < ld && col < td)
  {
    C[IDX2CKernel(row,col,td)] = A[IDX2CKernel(row,col,td)] * B;
  }
}

//Division
__global__ void MatDivide(float* A, float* B, float* C, int ld, int td)
{
  int col = (blockIdx.y * blockDim.y) + threadIdx.y;
  int row = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (row < ld && col < td)
  {
   C[IDX2CKernel(row,col,td)] = A[IDX2CKernel(row,col,td)] / B[IDX2CKernel(row,col,td)];
  }
}

__global__ void MatDivideConst(float* A, float B, float* C, int ld, int td)
{
  int col = (blockIdx.y * blockDim.y) + threadIdx.y;
  int row = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (row < ld && col < td)
  {
    C[IDX2CKernel(row,col,td)] = A[IDX2CKernel(row,col,td)] / B;
  }
}

//pow
__global__ void MatPow(float* A, float B, float* C, int ld, int td)
{
  int col = (blockIdx.y * blockDim.y) + threadIdx.y;
  int row = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (row < ld && col < td)
  {
    C[IDX2CKernel(row,col,td)] = powf(A[IDX2CKernel(row,col,td)],B);
  }
}

//CONVOLVE
__global__ void MatConvolve(float* A, float* B, float* C, int ald, int atd, int bld, int btd)
{
  int col = (blockIdx.y * blockDim.y) + threadIdx.y;
  int row = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (row < ald && col < atd)
  {
    C[IDX2CKernel(row,col,atd)] = 0.0;
    int radius = btd/2;
    for (int i = 0; i < btd; i++)
    {
      for (int j = 0; j < btd; j++)
      {
        int ii = i-radius;
        int jj = j-radius;
        if (row+ii >= 0 && row+ii < ald && col+jj >= 0 && col+jj < atd)
        {
          C[IDX2CKernel(row,col,atd)] += A[IDX2CKernel(row+ii,col+jj,atd)] * B[IDX2CKernel(i,j,btd)];
        }
      }
    }
  }

}
