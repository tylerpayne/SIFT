#include "hip/hip_runtime.h"
#include <utils//MatrixUtil.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include "kernels/MathKernels.cu"

#ifdef __cplusplus
  extern "C" {
#endif

hipStream_t _stream;
hipblasHandle_t _cublasHandle;
hipsolverHandle_t _cusolverHandle;

void cudaErrCheck(hipError_t stat)
{
  if (stat != hipSuccess)
  {
    printf("CUDA ERR\n%s\n",hipGetErrorString(stat));
  }
}

void cublasErrCheck(hipblasStatus_t stat)
{
  if (stat != HIPBLAS_STATUS_SUCCESS)
  {
    printf("\nCUBLAS ERR: %i\n",stat);
  }
}

void cusolverErrCheck(hipsolverStatus_t stat)
{
  if (stat != HIPSOLVER_STATUS_SUCCESS)
  {
    printf("\nCUSOLVER ERR: %i\n",stat);
  }
}

void freeCudaMatrixDeviceMemory(Matrix* mat)
{
  cudaErrCheck(hipFree(mat->devicePtr));
}

void freeCudaMatrixImpl(Matrix* m)
{
  printf("matfree\n");
  if (m->devicePtr != NULL)
  {
    freeCudaMatrixDeviceMemory(m);
  }
  free(m->hostPtr);
  free(m);
}

DLLEXPORT void copyDeviceToDeviceCudaMatrix(Matrix* A, Matrix* B)
{
    size_t size = sizeof(float)*A->shape.width*A->shape.height;
    cudaErrCheck(hipMemcpy(B->devicePtr,A->devicePtr,size,hipMemcpyDeviceToDevice));
}

DLLEXPORT void copyHostToDeviceCudaMatrix(Matrix* mat)
{
    size_t size = sizeof(float)*mat->shape.width*mat->shape.height;
    if (VERBOSITY > 3)
    {
      printf("\n\n##### GPU WARNING #####\n");
      printf("Copying from Host to Device");
      printf("\n###################\n\n");
    }
    cudaErrCheck(hipMemcpy(mat->devicePtr,mat->hostPtr,size,hipMemcpyHostToDevice));
    mat->isHostSide = 0;
}

DLLEXPORT void copyDeviceToHostCudaMatrix(Matrix* mat)
{
  size_t size = sizeof(float)*mat->shape.width*mat->shape.height;
  if (VERBOSITY > 3)
  {
    printf("\n\n### GPU WARNING ###\n");
    printf("Copying Device to Host");
    printf("\n###################\n\n");
  }
  cudaErrCheck(hipMemcpy(mat->hostPtr,mat->devicePtr,size,hipMemcpyDeviceToHost));
  mat->isHostSide = 1;
}

float getCudaMatrixElementImpl(Matrix* self, Point2 index)
{
  if (self->isHostSide)
  {
    return self->hostPtr[IDX2C(index,self->shape)];
  } else
  {
    return self->devicePtr[IDX2C(index,self->shape)];
  }
}

float* getCudaMatrixRegionImpl(Matrix* self, Rect rect)
{
  float* data;
  size_t s = sizeof(float)*rect.shape.width*rect.shape.height;
  if (self->isHostSide)
  {
    data = (float*)malloc(s);
  }
  else
  {
    cudaErrCheck(hipMalloc(&data,s));
  }

  for (int y = 0; y < rect.shape.height; y++)
  {
    for (int x = 0; x < rect.shape.width; x++)
    {
      Point2 tmp = {rect.origin.x,rect.origin.y};
      tmp.y += y;
      tmp.x += x;
      data[IDX2C(tmp,rect.shape)] = self->getElement(self,tmp);
    }
  }
  return data;
}

void setCudaMatrixElementImpl(Matrix* self, Point2 index, float x)
{
  if (self->isHostSide)
  {
    self->hostPtr[IDX2C(index,self->shape)] = x;
  }
  else
  {
    self->devicePtr[IDX2C(index,self->shape)] = x;
  }
}

void setCudaMatrixRegionImpl(Matrix* self, Rect rect, float* data)
{
  int counter = 0;
  for (int i = rect.origin.y; i< rect.origin.y+rect.shape.height; i++)
  {
    for (int j = rect.origin.x; j < rect.origin.x+rect.shape.width; j++)
    {
      Point2 idx = {j,i};
      self->setElement(self,idx,data[counter]);
      counter++;
    }
  }
}

Matrix* newEmptyCudaMatrixImpl(Shape shape)
{
  Matrix* m = (Matrix*)malloc(sizeof(Matrix));
  float* h_data = (float*)malloc(sizeof(float)*shape.height*shape.width);
  memset(h_data,0,sizeof(float)*shape.height*shape.width);
  float* d_data;
  cudaErrCheck(hipMalloc(&d_data,sizeof(float)*shape.height*shape.width));

  m->shape = shape;
  m->hostPtr = h_data;
  m->devicePtr = d_data;
  m->isHostSide = 1;
  m->T = HIPBLAS_OP_N;
  m->getElement = getCudaMatrixElementImpl;
  m->getRegion = getCudaMatrixRegionImpl;
  m->setElement = setCudaMatrixElementImpl;
  m->setRegion = setCudaMatrixRegionImpl;
  m->free = freeCudaMatrixImpl;

  return m;
}

Matrix* newCudaMatrixImpl(float* data, Shape shape)
{
  Matrix* m = newEmptyCudaMatrixImpl(shape);
  free(m->hostPtr);
  m->hostPtr = data;
  return m;
}
//############ BEGIN MATH FUNCS ##################
//ADDITION
void addCudaMatrixImpl(MatrixUtil* self, Matrix* A, Matrix* B, Matrix* C)
{
  if (A->isHostSide)
  {
    copyHostToDeviceCudaMatrix(A);
  }
  if (B->isHostSide)
  {
    copyHostToDeviceCudaMatrix(B);
  }
  if (C->isHostSide)
  {
    copyHostToDeviceCudaMatrix(C);
  }
  copyDeviceToDeviceCudaMatrix(B,C);
  float a = 1;
  cublasErrCheck(hipblasSaxpy(_cublasHandle,A->shape.height*A->shape.width,&a,A->devicePtr,1,C->devicePtr,1));
}

//SUBTRACTION
void subtractCudaMatrixImpl(MatrixUtil* self, Matrix* A, Matrix* B, Matrix* C)
{
  if (A->isHostSide)
  {
    copyHostToDeviceCudaMatrix(A);
  }
  if (B->isHostSide)
  {
    copyHostToDeviceCudaMatrix(B);
  }
  if (C->isHostSide)
  {
    copyHostToDeviceCudaMatrix(C);
  }
  copyDeviceToDeviceCudaMatrix(A,C);
  float a = -1;
  cublasErrCheck(hipblasSaxpy(_cublasHandle,A->shape.height*A->shape.width,&a,B->devicePtr,1,C->devicePtr,1));
}

//MULTIPLYCONST
void multiplyConstCudaMatrixImpl(MatrixUtil* self, Matrix* A, float b, Matrix* C)
{
  if (A->isHostSide)
  {
    copyHostToDeviceCudaMatrix(A);
  }
  if (C->isHostSide)
  {
    copyHostToDeviceCudaMatrix(C);
  }
  cublasErrCheck(hipblasSaxpy(_cublasHandle,A->shape.height*A->shape.width,&b,A->devicePtr,1,C->devicePtr,1));
}

//MIN
int* minRowsCudaMatrixImpl(MatrixUtil* self, Matrix* A)
{
  if (A->isHostSide)
  {
    copyHostToDeviceCudaMatrix(A);
  }

  int *idx = (int*)malloc(sizeof(int)*A->shape.height);
  for (int i = 0; i < A->shape.height; i++)
  {
    cublasErrCheck(hipblasIsamin(_cublasHandle, A->shape.width,
                            A->devicePtr+(i*A->shape.width), 1, idx+i));
    idx[i] -= 1;
  }
  return idx;
}

float maxValCudaMatrixImpl(MatrixUtil* self, Matrix* A)
{
  int c = self->maxIdx(self,A);
  Point2 idx = C2IDX(c,A->shape);
  return A->getElement(A,idx);
}

int maxIdxCudaMatrixImpl(MatrixUtil* self, Matrix* A)
{
  if (A->isHostSide)
  {
    copyHostToDeviceCudaMatrix(A);
  }

  int idx = 0;
  cublasErrCheck(hipblasIsamax(_cublasHandle, A->shape.height*A->shape.width,
                            A->devicePtr, 1, &idx));
  idx -= 1;
  return idx;
}

void powCudaMatrixImpl(MatrixUtil* self, Matrix* A, float k, Matrix* C)
{
  if (A->isHostSide)
  {
    copyHostToDeviceCudaMatrix(A);
  }
  if (C->isHostSide)
  {
    copyHostToDeviceCudaMatrix(C);
  }
  int bdimX = fmin(1024,A->shape.width*A->shape.height);
  dim3 bdim(bdimX);
  dim3 gdim(A->shape.width*A->shape.height/bdimX + 1);
  PowMatrixKernel<<<gdim,bdim,0,_stream>>>(A->devicePtr,k,C->devicePtr,A->shape.height,A->shape.width);
}

void transposeCudaMatrixImpl(MatrixUtil* self, Matrix* A, Matrix* C)
{
  if (A->isHostSide)
  {
    copyHostToDeviceCudaMatrix(A);
  }
  if (C->isHostSide)
  {
    copyHostToDeviceCudaMatrix(C);
  }
  int bdimX = fmin(32,A->shape.width);
  int bdimY = fmin(32,A->shape.height);
  dim3 bdim(bdimX,bdimY);
  dim3 gdim(A->shape.height/bdimX+1,A->shape.width/bdimY + 1);
  TransposeMatrixKernel<<<gdim,bdim,0,_stream>>>(A->devicePtr,C->devicePtr,A->shape.height,A->shape.width);
}

float distanceCudaMatrixImpl(MatrixUtil* self, Matrix* A, Matrix* B)
{
  if (A->isHostSide)
  {
    copyHostToDeviceCudaMatrix(A);
  }
  if (B->isHostSide)
  {
    copyHostToDeviceCudaMatrix(B);
  }

  Matrix* C = self->newEmptyMatrix(A->shape);

  self->subtract(self,A,B,C);

  float retval;
  cublasErrCheck(hipblasSnrm2(_cublasHandle,A->shape.height*A->shape.width,C->devicePtr,1,&retval));
  return retval;
}

//DOT
void dotCudaMatrixImpl(MatrixUtil* self, Matrix* A, Matrix* B, Matrix* C)
{
  if (A->isHostSide)
  {
    copyHostToDeviceCudaMatrix(A);
  }
  if (B->isHostSide)
  {
    copyHostToDeviceCudaMatrix(B);
  }
  if (C->isHostSide)
  {
    copyHostToDeviceCudaMatrix(C);
  }
  float alpha = 1;
  float beta = 0;
  int lda, tda, tdb;
  hipblasOperation_t opA, opB;
  if (A->T)
  {
    opA = HIPBLAS_OP_T;
    lda = A->shape.width;
    tda = A->shape.height;
  } else
  {
    opA = HIPBLAS_OP_N;
    lda = A->shape.height;
    tda = A->shape.width;
  }
  if (B->T)
  {
    opB = HIPBLAS_OP_T;
    tdb = B->shape.height;
  } else
  {
    opB = HIPBLAS_OP_N;
    tdb = B->shape.width;
  }
  cublasErrCheck(hipblasSgemm(_cublasHandle,
                           opA, opB,
                           lda, tdb, tda,
                           &alpha,
                           A->devicePtr, A->shape.height,
                           B->devicePtr, B->shape.height,
                           &beta,
                           C->devicePtr, C->shape.height));
}

void makeCrossMatrixImpl(MatrixUtil* self, Matrix* A, Matrix* Ax)
{
  if (A->isHostSide)
  {
    copyHostToDeviceCudaMatrix(A);
  }
  if (Ax->isHostSide)
  {
    copyHostToDeviceCudaMatrix(Ax);
  }
  Cross3X3MatrixKernel<<<1,1,0,_stream>>>(A->devicePtr,Ax->devicePtr);
}

//CROSS
void cross3X3MatrixImpl(MatrixUtil* self, Matrix* A, Matrix* B, Matrix* C)
{
  if (A->isHostSide)
  {
    copyHostToDeviceCudaMatrix(A);
  }
  if (B->isHostSide)
  {
    copyHostToDeviceCudaMatrix(B);
  }
  if (C->isHostSide)
  {
    copyHostToDeviceCudaMatrix(C);
  }
  Shape shape = {3,3};
  Matrix* Bx = self->newEmptyMatrix(shape);
  self->makeCrossMatrix(self,B,Bx);
  float alpha = 1;
  float beta = 0;
  cublasErrCheck(hipblasSgemv(_cublasHandle, HIPBLAS_OP_N, 3, 3, &alpha, Bx->devicePtr, 3, A->devicePtr, 1, &beta, C->devicePtr, 1));
  Bx->free(Bx);
}

void featureDistanceCudaMatrixImpl(MatrixUtil* self, Matrix* A, Matrix* B, Matrix* C)
{
  if (A->isHostSide)
  {
    copyHostToDeviceCudaMatrix(A);
  }
  if (B->isHostSide)
  {
    copyHostToDeviceCudaMatrix(B);
  }
  if (C->isHostSide)
  {
    copyHostToDeviceCudaMatrix(C);
  }
  int bdimX = fmin(32,B->shape.height);
  int bdimY = fmin(32,A->shape.height);
  dim3 bdim(bdimX,bdimY);
  dim3 gdim(B->shape.height/bdimX + 1,A->shape.height/bdimY + 1);
  FeatureDistanceMatrixKernel<<<gdim,bdim,0,_stream>>>(A->devicePtr,A->shape.height,B->devicePtr,B->shape.height,C->devicePtr,A->shape.width);
}

//inv
void invCudaMatrixImpl(MatrixUtil* self, Matrix* A, Matrix* Ainv)
{
  if (A->isHostSide)
  {
    copyHostToDeviceCudaMatrix(A);
  }
  if (Ainv->isHostSide)
  {
    copyHostToDeviceCudaMatrix(Ainv);
  }

  float* Acopy;
  size_t size = sizeof(float)*A->shape.height*A->shape.width;
  cudaErrCheck(hipMalloc(&Acopy,size));
  cudaErrCheck(hipMemcpy(Acopy,A->devicePtr,size,hipMemcpyDeviceToDevice));

  int n = A->shape.height;
  int Lwork;

  //GET BUFFER size
  cusolverErrCheck(hipsolverDnSgetrf_bufferSize(_cusolverHandle,
                      n,
                      n,
                      Acopy,
                      n,
                      &Lwork));

  //Create Workspace
  float* workspace;
  cudaErrCheck(hipMalloc(&workspace,Lwork));

  //Prepare LU decomposition
  int* devIpiv;
  cudaErrCheck(hipMalloc(&devIpiv,sizeof(int)*n));

  int* devInfo;
  cudaErrCheck(hipMalloc(&devInfo,sizeof(int)));

  //DECOMPOSE
  cusolverErrCheck(hipsolverDnSgetrf(_cusolverHandle,
           n,
           n,
           Acopy,
           n,
           workspace,
           devIpiv,
           devInfo));

  if (VERBOSITY > 3)
  {
    int* h_info = (int*)malloc(sizeof(int));
    cudaErrCheck(hipMemcpy(h_info,devInfo,sizeof(int),hipMemcpyDeviceToHost));
    printf("LU DECOMPOSITION INFO: %i\n",h_info[0]);
  }

  //right hand sides
  float B[] = {1,0,0,0,1,0,0,0,1};
  float *d_B;
  cudaErrCheck(hipMalloc(&d_B,size));
  cudaErrCheck(hipMemcpy(d_B,&B,size,hipMemcpyHostToDevice));

  //solve
  cusolverErrCheck(hipsolverDnSgetrs(_cusolverHandle,
           HIPBLAS_OP_N,
           n,
           n,
           Acopy,
           n,
           devIpiv,
           d_B,
           n,
           devInfo ));

  if (VERBOSITY > 3)
  {
    int* h_info = (int*)malloc(sizeof(int));
    cudaErrCheck(hipMemcpy(h_info,devInfo,sizeof(int),hipMemcpyDeviceToHost));
    printf("SOLVE INFO: %i\n",h_info[0]);
  }

  cudaErrCheck(hipMemcpy(Ainv->devicePtr,d_B,size,hipMemcpyDeviceToDevice));
  cudaErrCheck(hipFree(workspace));
  cudaErrCheck(hipFree(devIpiv));
  cudaErrCheck(hipFree(devInfo));
  cudaErrCheck(hipFree(Acopy));
  cudaErrCheck(hipFree(d_B));
}

void copyCudaMatrixImpl(MatrixUtil* self, Matrix* A, Matrix* B, Rect size, Point2 Aidx, Point2 Bidx)
{
  if (A->isHostSide)
  {
    copyHostToDeviceCudaMatrix(A);
  }
  if (B->isHostSide)
  {
    copyHostToDeviceCudaMatrix(B);
  }
  int bdimX = fmin(32,size.shape.width);
  int bdimY = fmin(32,size.shape.height);
  dim3 bdim(bdimX,bdimY);
  dim3 gdim(size.shape.width/bdimX + 1,size.shape.height/bdimY + 1);
  CopyMatrixKernel<<<gdim,bdim,0,_stream>>>(A->devicePtr,A->shape.height,A->shape.width,Aidx,B->devicePtr,B->shape.height,B->shape.width,Bidx,size);
}

void SetCUDAMatrixUtilStream(hipStream_t stream)
{
    _stream = stream;
    hipblasSetStream(_cublasHandle, stream);
    hipsolverSetStream(_cusolverHandle, stream);
}

void InitCUDAHandles()
{
  hipblasHandle_t cublasHandle;
  hipblasCreate(&cublasHandle);
  _cublasHandle = cublasHandle;

  hipsolverHandle_t cusolverHandle;
  hipsolverDnCreate(&cusolverHandle);
  _cusolverHandle = cusolverHandle;

  hipStream_t stream;
  hipStreamCreate(&stream);
  _stream = stream;
  SetCUDAMatrixUtilStream(stream);
}

void pprintCudaMatrixImpl(MatrixUtil* self, Matrix* A, char* label)
{
  printf("\n\n################################################");
  printf("\n%s:\n\n",label);
  if (!A->isHostSide)
  {
    copyDeviceToHostCudaMatrix(A);
  }
  for (int i = 0; i < A->shape.height; i++)
  {
    for (int j = 0;j< A->shape.width;j++)
    {
      Point2 idx = {j,i};
      printf("[ %f ]",A->getElement(A,idx));
    }
    //printf("|  %f",y->getElement(y,i,0));
    printf("\n");
  }
  printf("\n################################################\n\n");
}


  DLLEXPORT MatrixUtil* GetMatrixUtil()
{
  MatrixUtil* cudaMatrixUtil = (MatrixUtil*)malloc(sizeof(MatrixUtil));
  InitCUDAHandles();

  cudaMatrixUtil->newEmptyMatrix = newEmptyCudaMatrixImpl;
  cudaMatrixUtil->newMatrix = newCudaMatrixImpl;
  cudaMatrixUtil->copy = copyCudaMatrixImpl;
  cudaMatrixUtil->pprint = pprintCudaMatrixImpl;

  cudaMatrixUtil->add = addCudaMatrixImpl;
  cudaMatrixUtil->subtract = subtractCudaMatrixImpl;
  cudaMatrixUtil->dot = dotCudaMatrixImpl;
  cudaMatrixUtil->multiplyConst = multiplyConstCudaMatrixImpl;
  cudaMatrixUtil->distance = distanceCudaMatrixImpl;
  cudaMatrixUtil->makeCrossMatrix = makeCrossMatrixImpl;
  cudaMatrixUtil->cross = cross3X3MatrixImpl;
  cudaMatrixUtil->inv = invCudaMatrixImpl;
  cudaMatrixUtil->maxIdx = maxIdxCudaMatrixImpl;
  cudaMatrixUtil->maxVal = maxValCudaMatrixImpl;
  cudaMatrixUtil->minRows = minRowsCudaMatrixImpl;
  cudaMatrixUtil->pow = powCudaMatrixImpl;
  cudaMatrixUtil->featureDistance = featureDistanceCudaMatrixImpl;
  cudaMatrixUtil->transpose = transposeCudaMatrixImpl;
/*
  cudaMatrixUtil->multiply = multiplyCudaMatrixImpl;
  cudaMatrixUtil->divide = divideCudaMatrixImpl;
  cudaMatrixUtil->divideConst = divideConstCudaMatrixImpl;
  cudaMatrixUtil->pow = powCudaMatrixImpl;
  cudaMatrixUtil->convolve = convolveCudaMatrixImpl;
  cudaMatrixUtil->sqrt = sqrtCudaMatrixImpl;
  cudaMatrixUtil->exp = expCudaMatrixImpl;
  cudaMatrixUtil->log = logCudaMatrixImpl;
  cudaMatrixUtil->arctan = arctanCudaMatrixImpl;



  cudaMatrixUtil->solve = solveCudaMatrixImpl;
  cudaMatrixUtil->lstsq = lstsqCudaMatrixImpl;

  //cudaMatrixUtil->ceil = ceilCudaMatrixImpl;
  //cudaMatrixUtil->floor = floorCudaMatrixImpl;
  //cudaMatrixUtil->abs = absCudaMatrixImpl;
  cudaMatrixUtil->isEqual = isEqualCudaMatrixImpl;
  cudaMatrixUtil->convolve = convolveCudaMatrixImpl;*/

  return cudaMatrixUtil;
}
#ifdef __cplusplus
  }
#endif
